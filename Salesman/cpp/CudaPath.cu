#include "hip/hip_runtime.h"
#include "CudaPath.cuh"
#include <time.h>
#include <stdexcept>
#include <math.h>
#include <hip/hip_runtime.h>
#include <>
#include <limits>

CudaPath::CudaPath(std::vector<int> inputPoints) : Traveler(inputPoints){}

__global__
void cudaCopyNormDists(float* distMat, float* distVec, int * setIndizes,  int index, const int nPoints) {

	for (int i = 0; i < nPoints; i++) {
		distVec[i] = std::exp(distMat[index * nPoints + i] / -100);
	}

	for (int i = 0; i < nPoints; i++) {
		distVec[setIndizes[i]] = 0.0;
	}
}


__global__
void cudaSoftMax(float * distances, int distSize) {
	
	float total = 0;

	// sum up all distances
	for (int i = 0; i < distSize; i++) {
		total += distances[i];
	}

	// devide by total sum
	for (int i = 0; i < distSize; i++) {
		distances[i] = distances[i] / total;
	}
}

__global__
void cudaChooseNext(float* distances, float rand, int * index) {
	
	double cumTotal = 0;
	// index[0] = -1;

	do {
		index[0]++;
		cumTotal += distances[index[0]];
	} while (cumTotal < rand);
}

__global__
void particleGoBrrt(float* distMat, float* distances, int* setIndizes, float* rolls, float* totalDistances, const int nPoints) {
	
	int startIndex = 0;
	int threadBlockIdx = blockIdx.x * blockDim.x + threadIdx.x;

	totalDistances[threadBlockIdx] = 0.0;

	for (int i = 1; i < nPoints; i++) {

		// fill distances from startIndex Point with distances to other points
		for (int j = 0; j < nPoints; j++) {
			distances[threadBlockIdx*nPoints + j] = std::exp(distMat[startIndex * j + nPoints] / -100);
		}

		// set already used points to 0
		for (int j = 0; j < nPoints; j++) {
			distances[threadBlockIdx*nPoints + setIndizes[threadBlockIdx * nPoints + j]] = 0.0;
		}

		// calculate cumulative value
		float total = 0.0;
		for (int j = 0; j < nPoints; j++) {
			total += distances[threadBlockIdx * nPoints + j];
		}

		// devide each entry by total sum
		for (int j = 0; j < nPoints; j++) {
			distances[threadBlockIdx * nPoints + j] = distances[threadBlockIdx * nPoints + j] / total;
		}

		// select random
		float cumTotal = 0.0;
		int nextIndex = 0;
		do {
			nextIndex++;
			cumTotal += distances[threadBlockIdx * nPoints + nextIndex];
		} while ((cumTotal < rolls[threadBlockIdx * nPoints + i]) && (nextIndex < nPoints-1));

		// add distance to total distance
		totalDistances[threadBlockIdx] += distMat[startIndex * nPoints + nextIndex];

		// handle indizes
		startIndex = nextIndex;
		setIndizes[threadBlockIdx * nPoints + i] = nextIndex;
	}

	// path back to 0
	totalDistances[threadBlockIdx] += distMat[startIndex * nPoints];

}

__global__
void addTwo(int * x, int * y, int * z) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	z[i] = x[i] + y[i];
}

void testBasicStuff() {

	int N = 1 << 10;
	int* x, * y, * z, * d_x, * d_y, * d_z;

	x = (int*)malloc(N * sizeof(int));
	y = (int*)malloc(N * sizeof(int));
	z = (int*)malloc(N * sizeof(int));

	for (int i = 0; i < N; i++) {
		x[i] = 1;
		y[i] = 2;
	}

	// allocate memory

	hipMalloc(&d_x, N * sizeof(int));
	hipMalloc(&d_y, N * sizeof(int));
	hipMalloc(&d_z, N * sizeof(int));

	hipMemcpy(d_x, x, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_z, z, N * sizeof(int), hipMemcpyHostToDevice);

	addTwo << <(N + 255 / 256), 256 >> > (d_x, d_y, d_z);

	hipMemcpy(z, d_z, N * sizeof(int), hipMemcpyDeviceToHost);

	double maxError = 0.0;
	for (int i = 0; i < N; i++) {
		maxError = std::max(maxError, std::abs((double)z[i] - 3.0));
	}

	// free memory
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);

	free(x);
	free(y);
	free(z);
}

void CudaPath::runOneParticle(
		const std::vector<double>& distMat,
		std::default_random_engine & generator,
		std::vector<int> & travelIndizes,
		double & travelDist) {

	std::unordered_set<int> chosenIndizes;
	// start search from point 0
	int currentIndex = 0;
	chosenIndizes.insert(currentIndex);

	// random number init
	// std::default_random_engine generator;
	std::uniform_real_distribution<double> dist(0.0, 1.0);

	travelDist = 0;

	// allocate distance Mat on graphics card
	float * distMatSimple, * d_distMatSimple;
	distMatSimple = (float*)malloc(nPoints * nPoints * sizeof(float));
	hipMalloc(&d_distMatSimple, nPoints * nPoints * sizeof(float));

	// copy distMat into distMatSimple
	for (int i = 0; i < nPoints * nPoints; i++) {
		distMatSimple[i] = (float)distMat[i];
	}

	// copy to graphics card
	hipMemcpy(d_distMatSimple, distMatSimple, nPoints * nPoints * sizeof(float), hipMemcpyHostToDevice);

	// initialize indizes already set
	int* setIndizes, * d_setIndizes;
	setIndizes = (int*)malloc(nPoints * sizeof(int));
	hipMalloc(&d_setIndizes, nPoints * sizeof(int));

	for (int i = 0; i < nPoints; i++) {
		setIndizes[i] = 0;
	}

	hipMemcpy(d_setIndizes, setIndizes, nPoints * sizeof(int), hipMemcpyHostToDevice);

	for (int i = 1; i < nPoints; i++) {
		// init distances, set entries already chosen to 0

		// init distances, allocate memory space
		float* distsFromPoint, * d_dists;
		distsFromPoint = (float*)malloc(this->nPoints * sizeof(float));
		hipMalloc(&d_dists, nPoints * sizeof(float));

		cudaCopyNormDists<<<1,1>>>(d_distMatSimple, d_dists, d_setIndizes, i, nPoints);
		
		hipMemcpy(distsFromPoint, d_dists, nPoints * sizeof(float), hipMemcpyDeviceToHost);

		cudaSoftMax<<<1,1>>>(d_dists, nPoints);

		hipMemcpy(distsFromPoint, d_dists, nPoints * sizeof(float), hipMemcpyDeviceToHost);

		float roll = dist(generator);

		int * index, * d_index;

		index = (int*)malloc(sizeof(int));
		index[0] = -1;
		hipMalloc(&d_index, sizeof(int));
		hipMemcpy(d_index, index, sizeof(int), hipMemcpyHostToDevice);
		
		cudaChooseNext<<<1,1>>>(d_dists, roll, d_index);

		hipMemcpy(index, d_index, sizeof(int), hipMemcpyDeviceToHost);

		travelDist += distMat[currentIndex * nPoints + index[0]];
		
		chosenIndizes.insert(index[0]);
		travelIndizes.push_back(index[0]);
		setIndizes[i] = index[0];
		hipMemcpy(d_setIndizes, setIndizes, nPoints * sizeof(int), hipMemcpyHostToDevice);


		currentIndex = index[0];

		// Free memory
		hipFree(d_index);
		hipFree(d_dists);

		free(distsFromPoint);
		free(index);
	}

	travelDist += distMat[currentIndex];


}

void CudaPath::runParticles() {

	if (searched) {
		throw std::runtime_error("Reset before running new permutation run");
	}

	// init timing
	time_t start;
	time(&start);

	// create the distance matrix
	std::vector<double> distMat = this->createDistMat(this->points);

	// parameters for number of threads
	int threadBlocks = 3000;
	int threads = 256;
	int nParticles = threadBlocks * threads;

	// allocate memory and init distance matrix
	float* distMatSimple, * d_distMatSimple;
	distMatSimple = (float*)malloc(nPoints * nPoints * sizeof(float));
	hipMalloc(&d_distMatSimple, nPoints * nPoints * sizeof(float));

	// copy distMat into distMatSimple TODO if time rewrite for float
	for (int i = 0; i < nPoints * nPoints; i++) {
		distMatSimple[i] = (float)distMat[i];
	}

	// init array containing distances from current point
	float* distances, * d_distances;
	distances = (float*)malloc(nPoints * nParticles * sizeof(float));
	hipMalloc(&d_distances, nPoints * nParticles * sizeof(float));

	// init random rolls
	std::default_random_engine generator;
	std::uniform_real_distribution<double> dist(0.0, 1.0);

	float* rolls, * d_rolls;
	rolls = (float*)malloc(nParticles * nPoints * sizeof(float));
	hipMalloc(&d_rolls, nParticles * nPoints * sizeof(float));

	for (int i = 0; i < nPoints * nParticles; i++) {
		rolls[i] = dist(generator);
	}

	// array containing set indizes per particle

	int* setIndizes, * d_setIndizes;
	setIndizes = (int*)malloc(nPoints * nParticles * sizeof(int));
	hipMalloc(&d_setIndizes, nPoints * nParticles * sizeof(int));

	for (int i = 0; i < nPoints * nParticles; i++) {
		setIndizes[i] = 0;
	}

	// array containing total travel distance
	float* totalDistances, * d_totalDistances;
	totalDistances = (float*)malloc(nParticles * sizeof(float));
	hipMalloc(&d_totalDistances, nParticles * sizeof(float));

	// copy from host to device
	hipMemcpy(d_distMatSimple, distMatSimple, nPoints * nPoints * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_rolls, rolls, nParticles * nPoints * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_setIndizes, setIndizes, nParticles * nPoints * sizeof(int), hipMemcpyHostToDevice);
	


	particleGoBrrt << <threadBlocks, threads>> > (d_distMatSimple, d_distances, d_setIndizes, d_rolls, d_totalDistances, nPoints);

	// copy back results
	hipMemcpy(setIndizes, d_setIndizes, nPoints * nParticles * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(totalDistances, d_totalDistances, nParticles * sizeof(float), hipMemcpyDeviceToHost);

	// for debug purpose
	hipMemcpy(distances, d_distances, nPoints * nParticles * sizeof(int), hipMemcpyDeviceToHost);

	// find shortest total distance travelled by a particle
	int bestParticle;
	float bestDistance = std::numeric_limits<float>::max();
	
	// iter defined out of loop for debugging
	int iter;
	for (iter = 0; iter < nParticles; iter++) {
		if (bestDistance > totalDistances[iter]) {
			bestDistance = totalDistances[iter];
			bestParticle = iter;
			timeToFindBest = difftime(time(NULL), start);
		}
	}

	this->bestDist = (double)bestDistance;

	// copy indizes in result field

	for (int i = 1; i < nPoints; i++) {
		this->bestPathIndizes.push_back(setIndizes[bestParticle * nPoints + i]);
	}

	// free memory
	free(setIndizes);
	free(rolls);
	free(distMatSimple);
	free(distances);

	hipFree(d_setIndizes);
	hipFree(d_rolls);
	hipFree(d_distMatSimple);
	hipFree(d_distances);

	searched = true;
	timeToFinish = difftime(time(NULL), start);
}